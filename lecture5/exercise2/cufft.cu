#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hipfft/hipfft.h>
#include "../../cuda_error_check.h"

using namespace std;

#define CUFFT_ASSERT(x) if ((x) != HIPFFT_SUCCESS) { \
    cout << "Error at " << __FILE__ << ":" << __LINE__ << endl; \
    exit(1); \
}

void initialize_x(double *x, int n) {
    for (int i = 0; i < n; i++) {
        x[i] = i * (1.0 / double(n)) * (2 * M_PI);
    }
}

void initialize_f(hipfftDoubleComplex *f, double *x, int n) {
    for (int i = 0; i < n; i++) {
        f[i].x = cos(x[i]);
    }
}

void init_k(double *k, int n) {
    for (int i = 0; i <= n/2; i++) {
        k[i] = i;
    }
    for (int i = n/2 + 1; i < n; i++) {
        k[i] = (i - n);
    }
}

void verify_u(hipfftDoubleComplex *u, double *x, int n) {
    for (int i = 0; i < n; i++) {
        double u_exact = cos(x[i]);
        double u_approx = u[i].x; // -cos(x[i]);
        if (fabs(u_exact + u_approx) > 1e-6) {
            cout << "Verification failed at i = " << i << endl;
            cout << "Exact: " << u_exact << " Approx: " << u_approx << endl;
            exit(1);
        }
    }
    cout << "Verification passed!" << endl;
}

// ------------------ Main ------------------
int main(int argc, char **argv) {
    int n = 1 << 28;
    double *x;
    hipfftDoubleComplex *f_hat, *u_hat;
    hipfftDoubleComplex *f, *u;

    gpuErrorCheck(hipMallocManaged(&x, n * sizeof(double)));
    gpuErrorCheck(hipMallocManaged(&f, n * sizeof(hipfftDoubleComplex)));
    gpuErrorCheck(hipMallocManaged(&f_hat, n * sizeof(hipfftDoubleComplex)));
    gpuErrorCheck(hipMallocManaged(&u, n * sizeof(hipfftDoubleComplex)));
    gpuErrorCheck(hipMallocManaged(&u_hat, n * sizeof(hipfftDoubleComplex)));

    initialize_x(x, n);
    initialize_f(f, x, n);

    hipfftHandle handle;
    CUFFT_ASSERT(hipfftPlan1d(&handle, n, HIPFFT_Z2Z, 1));

    // transform f to f_hat
    CUFFT_ASSERT(hipfftExecZ2Z(handle, f, f_hat, HIPFFT_FORWARD));
    hipDeviceSynchronize();

    // Initialize k vector
    double *k;
    gpuErrorCheck(hipMallocManaged(&k, n * sizeof(double)));
    init_k(k, n);

    for (int i = 1; i < n; i++) {
        u_hat[i].x = -f_hat[i].x / (k[i] * k[i] * n);
        u_hat[i].y = -f_hat[i].y / (k[i] * k[i] * n);
    }

    u_hat[0].x = f_hat[0].x;
    u_hat[0].y = f_hat[0].y;

    hipDeviceSynchronize();

    // transform u_hat to u
    CUFFT_ASSERT(hipfftExecZ2Z(handle, u_hat, u, HIPFFT_BACKWARD));
    hipDeviceSynchronize();

    // Verify u
    verify_u(u, x, n);

    // Free the memory
    gpuErrorCheck(hipFree(x));
    gpuErrorCheck(hipFree(f));
    gpuErrorCheck(hipFree(f_hat));
    gpuErrorCheck(hipFree(u));
    gpuErrorCheck(hipFree(u_hat));

    // Destroy handle
    CUFFT_ASSERT(hipfftDestroy(handle));

    return EXIT_SUCCESS;
}