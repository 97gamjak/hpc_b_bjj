#include <hipblas.h>
#include <stdio.h>
#include <stdlib.h>

#include <string>

#include "../../cuda_error_check.h"

int main(int argc, char *argv[]) {
    int n = 10000;

    if (argc > 1) {
        int tmp = atoi(argv[1]);
        if (tmp > 0) {
            n = tmp;
        }
    }

    printf("Using arrays with %d elements.\n", n);

    // hipSetDevice(0);

    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("ERROR: hipblasCreate failed. Error code: %d\n", status);
        exit(1);
    }

    // - - - - - -

    size_t bytes_A = (n * n) * sizeof(double);
    double *h_A = (double *)malloc(bytes_A);
    for (int j = 0; j < n; j++)
        for (int i = 0; i < n; i++)
            h_A[i + n * j] = i - j;

    double *d_A;
    gpuErrorCheck(hipMalloc(&d_A, bytes_A));
    gpuErrorCheck(hipMemcpy(d_A, h_A, bytes_A, hipMemcpyHostToDevice));

    // - - - - - - -

    size_t bytes_xy = n * sizeof(double);

    double *h_x = (double *)malloc(bytes_xy);
    for (int j = 0; j < n; j++)
        h_x[j] = j;

    double *d_x;
    gpuErrorCheck(hipMalloc(&d_x, bytes_xy));
    gpuErrorCheck(hipMemcpy(d_x, h_x, bytes_xy, hipMemcpyHostToDevice));

    double *h_y = (double *)malloc(bytes_xy);
    for (int j = 0; j < n; j++)
        h_y[j] = 0;

    double *d_y;
    gpuErrorCheck(hipMalloc(&d_y, bytes_xy));
    gpuErrorCheck(hipMemcpy(d_y, h_y, bytes_xy, hipMemcpyHostToDevice));

    // - - - - - - -

    double alpha = 1.0;
    double beta = 0.0;  // if beta=0 then d_y can be uninitialized
    hipblasDgemv(handle, HIPBLAS_OP_N, n, n, &alpha, d_A, n, d_x, 1, &beta, d_y, 1);

    // - - - - - - -

    gpuErrorCheck(hipMemcpy(h_y, d_y, bytes_xy, hipMemcpyDeviceToHost));

    free(h_A);
    free(h_x);
    free(h_y);
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);

    hipblasDestroy(handle);

    return EXIT_SUCCESS;
}