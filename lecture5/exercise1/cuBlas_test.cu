
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <hipblas.h>

void handle_error(hipError_t error_id, std::string msg) {
    if (error_id != hipSuccess) {
        printf("Encountered cuda error %d. Msg: \"%s\"\n", error_id, msg.c_str());
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char *argv[]) {
    int n = 10000;

    if (argc > 1) {
        int tmp = atoi(argv[1]);
        if (tmp > 0) {
            n = tmp;
        }
    }

    printf("Using arrays with %d elements.\n", n);

    // cudaSetDevice(0);

    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle); 
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("ERROR: hipblasCreate failed. Error code: %d\n", status);
        exit(1);
    }

    // - - - - - -

    size_t bytes_A = (n * n) * sizeof(double);
    double* h_A = (double*)malloc(bytes_A); 
    for(int j=0;j<n;j++)
        for(int i=0;i<n;i++) 
            h_A[i + n*j] = i-j;
    
    double *d_A;
    handle_error(hipMalloc(&d_A, bytes_A), "allocate d_A");
    handle_error(hipMemcpy(d_A, h_A, bytes_A, hipMemcpyHostToDevice), "copy h_A to device");

    // - - - - - - -

    size_t bytes_xy = n * sizeof(double);

    double *h_x = (double *)malloc(bytes_xy);
    for(int j=0; j<n; j++)
        h_x[j] = j;

    double *d_x;
    handle_error(hipMalloc(&d_x, bytes_xy), "allocate d_x");
    handle_error(hipMemcpy(d_x, h_x, bytes_xy, hipMemcpyHostToDevice), "copy h_x to device");

    double *h_y = (double *)malloc(bytes_xy);
    for(int j=0; j<n; j++)
        h_y[j] = 0;

    double *d_y;
    handle_error(hipMalloc(&d_y, bytes_xy), "allocate d_y");
    handle_error(hipMemcpy(d_y, h_y, bytes_xy, hipMemcpyHostToDevice), "copy h_y to device");

    // - - - - - - -

    double alpha = 1.0;
    double beta = 0.0; // if beta=0 then d_y can be uninitialized 
    hipblasDgemv(handle, HIPBLAS_OP_N, n, n, &alpha, d_A, n, d_x, 1, &beta, d_y, 1);

    // - - - - - - -

    handle_error(hipMemcpy(h_y, d_y, bytes_xy, hipMemcpyDeviceToHost), "copy d_y to host");

    free(h_A);
    free(h_x);
    free(h_y);
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);

    hipblasDestroy(handle);

    return EXIT_SUCCESS;
}