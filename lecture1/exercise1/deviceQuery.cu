
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

int main(int argc, char *argv[])
{
    int deviceCount;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }

    for (int dev = 0; dev < deviceCount; ++dev)
    {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        cout << "-------------------------------------------------------------" << endl;
        cout << "Device " << dev << ": \"" << deviceProp.name << "\"" << endl;
        cout << "\tCUDA Driver and Runtime version: " << deviceProp.major << "." << deviceProp.minor << endl;
        cout << "\tTotal amount of global memory: " << deviceProp.totalGlobalMem / 1e9 << " GB" << endl;
        cout << "\tMaximum number of threads per block: " << deviceProp.maxThreadsPerBlock << endl;
        cout << "\tUVA support: " << (deviceProp.unifiedAddressing ? "Yes" : "No") << endl;
        cout << "-------------------------------------------------------------" << endl;
        
    }
    return EXIT_SUCCESS;
}