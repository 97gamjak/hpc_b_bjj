
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

int main(int argc, char *argv[])
{
    int deviceCount;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (error_id != hipSuccess)
    {
        cout << "hipGetDeviceCount returned " << (int)error_id << endl << "-> " << hipGetErrorString(error_id) << endl;
        cout << "Result = FAIL" << endl;
        exit(EXIT_FAILURE);
    }

    for (int dev = 0; dev < deviceCount; ++dev)
    {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        cout << "-------------------------------------------------------------" << endl;
        cout << "Device " << dev << ": \"" << deviceProp.name << "\"" << endl;
        cout << "\tCUDA Driver and Runtime version: " << deviceProp.major << "." << deviceProp.minor << endl;
        cout << "\tTotal amount of global memory: " << deviceProp.totalGlobalMem / 1e9 << " GB" << endl;
        cout << "\tMaximum number of threads per block: " << deviceProp.maxThreadsPerBlock << endl;
        cout << "\tUVA support: " << (deviceProp.unifiedAddressing ? "Yes" : "No") << endl;
        cout << "-------------------------------------------------------------" << endl;
        
    }
    return EXIT_SUCCESS;
}