
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int main()
{
    int deviceCount;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    for (int dev = 0; dev < deviceCount; ++dev)
    {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        printf("\nDevice %ld: \"%s\"\n\t> Max gridsize: %ld\n\t> Max threadsPerBlock: %ld\n", dev, deviceProp.name, deviceProp.maxGridSize, deviceProp.maxThreadsPerBlock);
    }
    return 0;
}