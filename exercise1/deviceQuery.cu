
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int main()
{
    int deviceCount;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    for (int dev = 0; dev < deviceCount; ++dev)
    {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        printf("\nDevice %d: \"%s\"\n\t> Max gridsize: %d\n\t> Max threadsPerBlock: %d\n", dev, deviceProp.name, deviceProp.maxGridSize, deviceProp.maxThreadsPerBlock);
    }
    return 0;
}