
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
using namespace std;


__global__
void k_sum(long n, double* vec, double* result) {
    extern __shared__ float sdata[];
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // Load the data into shared memory
    sdata[tid] = (i < n) ? vec[i] : 0;
    __syncthreads();

    // Do the reduction in shared memory
    for(int s=blockDim.x/2; s>0; s>>=1) {
        if(tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if(tid == 0) {
        result[blockIdx.x] = sdata[0];
    }

    __syncthreads();
}

int main() {
    long n = 8*1024*1024; // must be a power of 2
    double h_result = 0.0;
    double *h_vec, *d_vec, *d_tmp;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipHostMalloc(&h_vec, sizeof(double)*n, hipHostMallocDefault);
    hipMalloc(&d_vec, sizeof(double)*n);
    hipMalloc(&d_tmp, sizeof(double)*(n/256));

    // Initialie vec and copy to GPU.
    for(long i=0;i<n;i++)
        h_vec[i] = 1.0/pow(double(i+1),2);
    hipMemcpy(d_vec, h_vec, sizeof(double)*n, hipMemcpyHostToDevice);

    // calculate number of blocks and threads per block
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    hipEventRecord(start, 0);
    while(numBlocks > 1) {
        k_sum<<<numBlocks, blockSize, blockSize*sizeof(double)>>>(n, d_vec, d_tmp);
        hipDeviceSynchronize();
        n = numBlocks;
        d_vec = d_tmp;
        numBlocks = (n + blockSize - 1) / blockSize;
    }
    hipEventRecord(stop, 0);

    k_sum<<<1, blockSize, blockSize*sizeof(double)>>>(n, d_vec, d_tmp);

    // Copy the result back to the host.
    hipMemcpy(&h_result, d_tmp, sizeof(double), hipMemcpyDeviceToHost);
    
    // Check the result.
    cout << "Result: " << h_result << endl;
    if(fabs(h_result - pow(M_PI,2)/6.0) < 1e-5) {
        cout << "Correct!" << endl;
    } else {
        cout << "The computed result does not match with the expected result ("
             << pow(M_PI,2)/6.0 << ")" << endl;
    }

    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);
    cout << time * 1e-3 << " s" << endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Free the memory.
    hipHostFree(h_vec);
    hipFree(d_vec);
    hipFree(d_tmp);
    return 0;
}

